﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void add(int* a, int* b, int* c, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {  
        c[index] = a[index] + b[index];
    }
}

int* dev_a;
int* dev_b;
int* dev_c;
int size;
hipStream_t stream;

extern "C" __declspec(dllexport) void create(int _size) {
    size = _size;  // 전역 변수에 사이즈 저장

    // GPU 메모리 할당
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    hipMalloc((void**)&dev_c, size * sizeof(int));

    // 스트림 생성
    hipStreamCreate(&stream);
}

extern "C" __declspec(dllexport) void copyToDevice(int* a, int* b) {
    // 비동기 전송 (CPU -> GPU)
    hipMemcpyAsync(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice, stream);
}
extern "C" __declspec(dllexport) void addArrays() {
    // 비동기 커널 실행
    int threadsPerBlock = 512;
    int numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    add <<< numBlocks, threadsPerBlock, 0, stream >>> (dev_a, dev_b, dev_c, size);
}

extern "C" __declspec(dllexport) void copyToHost(int* c) {
    // 비동기 전송 (GPU -> CPU)
    hipMemcpyAsync(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost, stream);
}

extern "C" __declspec(dllexport) void release() {
    // 스트림 작업 완료 기다림
    hipStreamSynchronize(stream);

    // 스트림과 메모리 해제
    hipStreamDestroy(stream);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}